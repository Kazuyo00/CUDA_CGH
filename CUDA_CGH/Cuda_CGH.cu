#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#define number 284
#define Wid 1920
#define Hgt 1080
#define tnum 1
#define device 0

#include <stdio.h>
#include <cmath>
#include <vector>
#include <sys/time.h>
#include <stdlib.h>
//#include <hip/hip_runtime.h>

#pragma pack(push,1)
typedef struct tagBITMAPFILEHEADER
{
	unsigned short bfType;
	int            bfSize;
	unsigned short bfReserved1;
	unsigned short bfReserved2;
	int            bf0ffBits;
}BITMAPFILEHEADER;

#pragma pack(pop)

typedef struct tagBITMAPINFOHEADER
{
	int             biSize;
	int			    biWid;
	int			    biHgt;
	unsigned short  biPlanes;
	unsigned short  biBitCount;
	int             biCompression;
	int             biSizeImage;
	int			    biXPelsPerMeter;
	int			    biYPelsPerMeter;
	int             biCirUsed;
	int             biCirImportant;
}BITMAPINFOHEADER;

typedef struct tagRGBQUAD
{
	unsigned char  rgbBlue;
	unsigned char  rgbGreen;
	unsigned char  rgbRed;
	unsigned char  rgbReserved;
}RGBQUAD;

typedef struct tagBITMAPINFO
{
	BITMAPINFOHEADER bmiHeader;
	RGBQUAD          bmiColors[1];
}BITMAPINFO;

/*時間計測*/
double gettimeofday_sec()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + tv.tv_usec * 1e-6;
}

/*カーネル関数*/
__global__ void holo_culculation(int *o_x, int *o_y, float *o_z, float *o_I){
	int i, j, k, adr;

	i = blockIdx.x*blockDim.x + threadIdx.x;
	j = blockIdx.y*blockDim.y + threadIdx.y;
	adr = i + j*Wid;

	float d_x, d_y, d_z, rr;
	float interval=10.5F;				//画素間隔
	float wave_len=0.633F;				//光波長
	float wave_num=2.0F*3.14159265F/wave_len;	//波数
	float kp=interval*wave_num;

	for(k=0; k<number; k++){
		d_x = ((float)j - o_x[k]) * ((float)j - o_x[k]);
		d_y = ((float)i - o_y[k]) * ((float)i - o_y[k]);
		d_z = o_z[k] * o_z[k];
		rr = sqrt(d_x + d_y + d_z);
		o_I[adr] = o_I[adr] + __cosf(kp*rr);
	}
}

unsigned char img[Hgt*Wid];
float I[Hgt*Wid];

int main()
{
	BITMAPFILEHEADER    BmpFileHeader;
	BITMAPINFOHEADER    BmpInfoHeader;
	RGBQUAD             RGBQuad[256];

	BmpFileHeader.bfType = 19778;
	BmpFileHeader.bfSize = 14 + 40 + 1024 + (256 * 256);
	BmpFileHeader.bfReserved1 = 0;
	BmpFileHeader.bfReserved2 = 0;
	BmpFileHeader.bf0ffBits = 14 + 40 + 1024;

	BmpInfoHeader.biSize = 40;
	BmpInfoHeader.biWid = Wid;
	BmpInfoHeader.biHgt = Hgt;
	BmpInfoHeader.biPlanes = 1;
	BmpInfoHeader.biBitCount = 8;
	BmpInfoHeader.biCompression = 0L;
	BmpInfoHeader.biSizeImage = 0L;
	BmpInfoHeader.biXPelsPerMeter = 0L;
	BmpInfoHeader.biYPelsPerMeter = 0L;
	BmpInfoHeader.biCirUsed = 0L;
	BmpInfoHeader.biCirImportant = 0L;

	int i, j, n;

	for (i = 0; i<256; i++){
		RGBQuad[i].rgbBlue = i;
		RGBQuad[i].rgbGreen = i;
		RGBQuad[i].rgbRed = i;
		RGBQuad[i].rgbReserved = 0;
	}

	hipSetDevice(device);

	FILE *fp;

	fp = fopen("cube284.3d", "rb");
	fread(&n, sizeof(int), 1, fp);

	int x[number], y[number], x1, y1, z1;
	float z[number];

	for (i = 0; i<number; i++){
		fread(&x1, sizeof(int), 1, fp);
		fread(&y1, sizeof(int), 1, fp);
		fread(&z1, sizeof(int), 1, fp);

		x[i] = x1 * 40 + Hgt / 2;
		y[i] = y1 * 40 + Wid / 2;
		z[i] = (float)z1 * 40 + 50000.0F;
	}

	fclose(fp);

	double starttime, endtime, time_tmp;

	for (i = 0; i < Hgt; i++){
		for (j = 0; j < Wid; j++){
			I[i * Wid +j] = 0.0;
		}
	}

	starttime=gettimeofday_sec();

	int *o_x, *o_y;
	float *o_z, *o_I;

	dim3 block(32,8,1); //スレッド数(ブロック分割)
	dim3 grid(ceil(Wid/block.x),ceil(Hgt/block.y),1); //ブロック数(グリッド分割)

	hipMalloc((void**)&o_x, number*sizeof(int));
	hipMalloc((void**)&o_y, number*sizeof(int));
	hipMalloc((void**)&o_z, number*sizeof(float));
	hipMalloc((void**)&o_I, Wid*Hgt*sizeof(float));

	hipMemcpy(o_x, x, number*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(o_y, y, number*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(o_z, z, number*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(o_I, I, Wid*Hgt*sizeof(float), hipMemcpyHostToDevice);

	holo_culculation<<< grid, block >>>(o_x, o_y, o_z, o_I);

	hipMemcpy(I, o_I, Wid*Hgt*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(o_x);
	hipFree(o_y);
	hipFree(o_z);
	hipFree(o_I);

	endtime=gettimeofday_sec();

	time_tmp = endtime-starttime;
	printf("%lf\n",time_tmp);

	time_tmp = 0.0;

	float max_tmp = 0.0, min_tmp = 0.0, mid_tmp = 0.0;

	max_tmp = I[0];
	min_tmp = I[0];

	for (i = 0; i < Hgt*Wid; i++){
		if (max_tmp <= I[i]){
			max_tmp = I[i];
//			printf("max i = %d\n",i);
		}

		else if (min_tmp > I[i]){
			min_tmp = I[i];
//			printf("min i = %d\n",i);
		}
	}

	mid_tmp = (max_tmp + min_tmp) * 0.5;

	printf("max = %lf\n", max_tmp);
	printf("min = %lf\n", min_tmp);
	printf("mid = %lf\n", mid_tmp);

	for (i = 0; i < Hgt*Wid; i++){
			img[i] = 0;
	}

	for (i = 0; i < Hgt*Wid; i++){

			if (I[i] <= mid_tmp){
				img[i] = 0;
			}

			else if (I[i] > mid_tmp){
				img[i] = 255;
			}
	}

	fp = fopen("CGH.bmp", "wb");

	fwrite(&BmpFileHeader, sizeof(BmpFileHeader), 1, fp);
	fwrite(&BmpInfoHeader, sizeof(BmpInfoHeader), 1, fp);
	fwrite(&RGBQuad[0], sizeof(RGBQuad[0]), 256, fp);
	fwrite(img, sizeof(unsigned char), Hgt * Wid, fp);

	fclose(fp);

	return 0;
}
